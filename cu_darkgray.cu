
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <chrono>
//#include <CImg.h>
//#define TIMERS

//using cimg_library::CImg;
using std::cout;
using std::cerr;
using std::endl;

/* Kernel for the device */
__global__ void rgb_gray(const int width, const int height,
        const unsigned char *inputImage,
        unsigned char *darkGrayImage) {
    int x;
    int y;

    // calculate the thread index for both x, y, by the use of the dimension
    // of the block the id of the current block and the id of the thread
    y = blockDim.y * blockIdx.y + threadIdx.y;
    x = blockDim.x * blockIdx.x + threadIdx.x;

    // check if we are out of bounds
    if ((y * width + x) > (width * height)) {
        return;
    }
    // do the transformation
    float grayPix = 0.0f;
    float r = static_cast<float>(inputImage[(y * width) + x]);
    float g = static_cast<float>(inputImage[(width * height) + (y * width) + x]);
    float b =
        static_cast<float>(inputImage[(2 * width * height) + (y * width) + x]);
    grayPix = ((0.3f * r) + (0.59f * g) + (0.11f * b));
    grayPix = (grayPix * 0.6f) + 0.5f;
    darkGrayImage[(y * width) + x] = static_cast<unsigned char>(grayPix);
}
// End Kernel
// Host
void darkGray(const int width, const int height, const int size, 
        const unsigned char *inputImage, unsigned char *darkGrayImage, 
        const int sizeOut) {
    unsigned char *inputImageDev;			// Input image on device
    unsigned char *darkGrayImageDev;		// Output image on device
    int size_image, outImageSize;			// Size of the image
    /* Find the size of the image */
    size_image = size * sizeof(*inputImage);
    outImageSize = sizeOut * sizeof(*darkGrayImage);

#ifdef TIMERS
    /* timer for input creation */
    std::chrono::time_point<std::chrono::system_clock> start, stop;
    start = std::chrono::system_clock::now();
#endif

    if (hipMalloc((void**)&inputImageDev, size_image) != hipSuccess) {
        cerr << "Cuda Malloc FAILED " << endl;
    }
    if (hipMalloc((void**)&darkGrayImageDev, outImageSize) != hipSuccess) {
        cerr << "Cuda Malloc FAILED " << endl;
    }
    
    hipMemset(darkGrayImageDev, 0 , outImageSize);

#ifdef TIMERS
    stop = std::chrono::system_clock::now();
    std::chrono::duration<double> elapsed = stop - start;
    cout << "DarkGray malloc: " << elapsed.count() << " sec." << endl;
    /*timer for input creation*/
    start = std::chrono::system_clock::now();
#endif

    //transfer image from  host to device
    if (hipMemcpy(inputImageDev, inputImage, size_image , hipMemcpyHostToDevice)!=hipSuccess){
        cerr << "Cuda MemCpy H2D FAILED " << endl;
    }

#ifdef TIMERS
    stop = std::chrono::system_clock::now();
    elapsed = stop - start;
    cout << "DarkGray H2D: " << elapsed.count() << " sec." << endl;

    /*timer for input creation*/
    start = std::chrono::system_clock::now();
#endif

    //find the width of the block
    int wBlock = static_cast<unsigned int>(ceil(width / static_cast<float>(32)));
    int hBlock = static_cast<unsigned int>(ceil(height / static_cast<float>(16)));

    //execution configuration
    dim3 dimGrid(wBlock,hBlock);//grid dimensions: (wBlock*hBlock) thread blocks
    dim3 dimBlock(32 , 16);//block dimensions: 32*16=512 threads per block

    //launch the kernel with dimGrid num of blocks and dimBlock num of threads eac
    rgb_gray<<<dimGrid, dimBlock>>>(width, height, inputImageDev,darkGrayImageDev);

    hipError_t err = hipGetLastError();

#ifdef TIMERS
    cudaDeviceSynchronize();

    stop = std::chrono::system_clock::now();
    elapsed = stop - start;
    cout<<std::fixed << "DarkGray kernel: " << elapsed.count() << " sec." << endl;

    /*timer for input creation*/
    start = std::chrono::system_clock::now();
#endif

    if (err != hipSuccess) 
        cerr << "Error: " << hipGetErrorString(err) << endl;

    if (hipMemcpy(darkGrayImage, darkGrayImageDev, outImageSize, hipMemcpyDeviceToHost)!=hipSuccess){
        cerr << "Cuda MemCpy D2H FAILED "<<endl;
    }
#ifdef TIMERS
    stop = std::chrono::system_clock::now();
    elapsed = stop - start;
    cout << "DarkGray D2H: " << elapsed.count() << " sec." << endl;

    /*timer for input creation*/
    start = std::chrono::system_clock::now();
#endif

    //clean up
    hipFree(inputImageDev);
    hipFree(darkGrayImageDev);

#ifdef TIMERS
    stop = std::chrono::system_clock::now();
    elapsed = stop - start;
    cout << "DarkGray Free: " << elapsed.count() << " sec." << endl;
#endif
}
